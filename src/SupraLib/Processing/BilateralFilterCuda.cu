#include "hip/hip_runtime.h"
// ================================================================================================
// 
// If not explicitly stated: Copyright (C) 2019, all rights reserved,
//      Rüdiger Göbl 
//		Email r.goebl@tum.de
//      Chair for Computer Aided Medical Procedures
//      Technische Universität München
//      Boltzmannstr. 3, 85748 Garching b. München, Germany
// 
// ================================================================================================

#include "BilateralFilterCuda.h"
#include "utilities/Buffer.h"

#include <thrust/transform.h>
#include <thrust/execution_policy.h>

using namespace std;

namespace supra
{
	namespace BilateralFilterCudaInternal
	{
		typedef BilateralFilterCuda::WorkType WorkType;

		// here the actual processing happens!

		template <typename InputType, typename OutputType>
		__global__ void processKernel(const InputType* inputImage, const vec3s size, 
									const vec3T<WorkType> sigmaSpatial, const vec3s filterSize,
									WorkType sigmaItensity, OutputType* outputImage)
		{
			size_t x = blockDim.x*blockIdx.x + threadIdx.x;
			size_t y = blockDim.y*blockIdx.y + threadIdx.y;
			size_t z = blockDim.z*blockIdx.z + threadIdx.z;
			
			size_t width = size.x;
			size_t height = size.y;
			size_t depth = size.z;

			// declare the shared memory with dynamic size
			extern __shared__ uint8_t smem[];

			// create the buffer objects
			// These wrap the image pointers and allow for easy indexing and caching (here for inputBuffer)
			CachedBuffer3<const InputType*, size_t> inputBuffer{
				inputImage, size, reinterpret_cast<InputType*>(smem), vec3s{blockDim.x, blockDim.y, blockDim.z},
				vec3s{blockDim.x*blockIdx.x, blockDim.y*blockIdx.y, blockDim.z*blockIdx.z } };
			Buffer3<OutputType*, size_t> outputBuffer{ outputImage, size };

			if (x < width && y < height && z < depth)
			{
				// Perform the computations for one output-pixel
				// -> loop over the spatial window
				vec3s indexStart{
					max(x, filterSize.x / 2) - filterSize.x / 2,
					max(y, filterSize.y / 2) - filterSize.y / 2,
					max(z, filterSize.z / 2) - filterSize.z / 2 };
				vec3s indexEnd{
					min(x + filterSize.x / 2, width - 1),
					min(y + filterSize.y / 2, height - 1),
					min(z + filterSize.z / 2, depth - 1) };
				WorkType accumulatedWeight = 0;
				WorkType valueFiltered = 0;
				WorkType valueCenter = inputBuffer[{x, y, z}];
				vec3s index{ indexStart };
				for (; index.z <= indexEnd.z; index.z++)
				{
					index.y = indexStart.y;
					for (; index.y <= indexEnd.y; index.y++)
					{
						index.x = indexStart.x;
						for (; index.x <= indexEnd.x; index.x++)
						{
							WorkType valueOffset = inputBuffer[index];
							// Compute the weight for this pixel in the window
							// in this case for the bilateral filter
							WorkType weight = 
								exp(
									// spatial term
									- (squ(index.x - x) / (2*squ(sigmaSpatial.x))
										+ squ(index.y - y) / (2*squ(sigmaSpatial.y))
										+ squ(index.z - z) / (2*squ(sigmaSpatial.z)))
									// intensity term
									- squ(valueOffset - valueCenter) / (2*squ(sigmaItensity))
								);
							// Update the result
							valueFiltered += weight * valueOffset;
							// remember used weights for later normalization
							accumulatedWeight += weight;
						}
					}
				}

				// normalize result w.r.t. employed weights
				if (accumulatedWeight != 0)
				{
					valueFiltered /= accumulatedWeight;
				}

				// Store the output pixel value.
				// Because this is templated, we need to cast from "WorkType" to "OutputType".
				// This should happen in a sane way, that is with clamping. There is a helper for that!
				outputBuffer[{x, y, z}] = clampCast<OutputType>(valueFiltered);
			}
		}
	}

	template <typename InputType, typename OutputType>
	shared_ptr<Container<OutputType> > BilateralFilterCuda::process(
		const shared_ptr<const Container<InputType>>& imageData, vec3s size, 
		const vec3T<WorkType>& sigmaSpatialPixels, WorkType sigmaItensity)
	{
		// here we prepare the buffers and call the cuda kernel

		size_t width = size.x;
		size_t height = size.y;
		size_t depth = size.z;

		// precompute parameters
		vec3s filterSize = static_cast<vec3s>(ceil(sigmaSpatialPixels * 2) * 2 + 1);

		// make sure the data is in cpu memory
		auto inImageData = imageData;
		if (!inImageData->isGPU() && !inImageData->isBoth())
		{
			inImageData = make_shared<Container<InputType> >(LocationGpu, *inImageData);
		}
		
		// prepare the output memory
		auto outImageData = make_shared<Container<OutputType> >(LocationGpu, inImageData->getStream(), width*height*depth);
		
		// call the kernel for the heavy-lifting
		dim3 blockSize(32, 4, 1);
		dim3 gridSize(
			static_cast<unsigned int>((size.x + blockSize.x - 1) / blockSize.x),
			static_cast<unsigned int>((size.y + blockSize.y - 1) / blockSize.y),
			static_cast<unsigned int>((size.z + blockSize.z - 1) / blockSize.z));
		// since we will use a caching buffer object inside the kernel, that places parts of the input 
		// in shared memory, we have to specify the size of that
		size_t sharedMemorySize = blockSize.x * blockSize.y * blockSize.z * sizeof(InputType);
		BilateralFilterCudaInternal::processKernel <<<gridSize, blockSize, sharedMemorySize, inImageData->getStream() >>> (
			inImageData->get(),
			size,
			sigmaSpatialPixels, 
			filterSize,
			sigmaItensity,
			outImageData->get());
		// check for cuda launch errors
		cudaSafeCall(hipPeekAtLastError());
		// You should NOT synchronize the device or the stream we are working on!!

		// return the result!
		return outImageData;
	}

	// We don't wish to have the template implementation in the header, to make compilation easier.
	// Because of this, we need to explicity instantiate the methods we will need.
	template
	shared_ptr<Container<uint8_t> > BilateralFilterCuda::process<int16_t, uint8_t>(
		const shared_ptr<const Container<int16_t> >& imageData, vec3s size,
		const vec3T<WorkType>& sigmaSpatialPixels, WorkType sigmaItensity);
	template
	shared_ptr<Container<uint8_t> > BilateralFilterCuda::process<float, uint8_t>(
		const shared_ptr<const Container<float> >& imageData, vec3s size,
		const vec3T<WorkType>& sigmaSpatialPixels, WorkType sigmaItensity);
	template
	shared_ptr<Container<uint8_t> > BilateralFilterCuda::process<uint8_t, uint8_t>(
		const shared_ptr<const Container<uint8_t> >& inImageData, vec3s size,
		const vec3T<WorkType>& sigmaSpatialPixels, WorkType sigmaItensity);
	template
	shared_ptr<Container<float> > BilateralFilterCuda::process<int16_t, float>(
		const shared_ptr<const Container<int16_t> >& inImageData, vec3s size,
		const vec3T<WorkType>& sigmaSpatialPixels, WorkType sigmaItensity);
	template
	shared_ptr<Container<float> > BilateralFilterCuda::process<float, float>(
		const shared_ptr<const Container<float> >& inImageData, vec3s size,
		const vec3T<WorkType>& sigmaSpatialPixels, WorkType sigmaItensity);
	template
	shared_ptr<Container<float> > BilateralFilterCuda::process<uint8_t, float>(
		const shared_ptr<const Container<uint8_t> >& inImageData, vec3s size,
		const vec3T<WorkType>& sigmaSpatialPixels, WorkType sigmaItensity);
	template
	shared_ptr<Container<int16_t> > BilateralFilterCuda::process<int16_t, int16_t>(
		const shared_ptr<const Container<int16_t> >& inImageData, vec3s size,
		const vec3T<WorkType>& sigmaSpatialPixels, WorkType sigmaItensity);
	template
	shared_ptr<Container<int16_t> > BilateralFilterCuda::process<float, int16_t>(
		const shared_ptr<const Container<float> >& inImageData, vec3s size,
		const vec3T<WorkType>& sigmaSpatialPixels, WorkType sigmaItensity);
	template
	shared_ptr<Container<int16_t> > BilateralFilterCuda::process<uint8_t, int16_t>(
		const shared_ptr<const Container<uint8_t> >& inImageData, vec3s size,
		const vec3T<WorkType>& sigmaSpatialPixels, WorkType sigmaItensity);
}