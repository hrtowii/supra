#include "hip/hip_runtime.h"
// ================================================================================================
// 
// If not explicitly stated: Copyright (C) 2017, all rights reserved,
//      Rüdiger Göbl 
//		Email r.goebl@tum.de
//      Chair for Computer Aided Medical Procedures
//      Technische Universität München
//      Boltzmannstr. 3, 85748 Garching b. München, Germany
// 
// ================================================================================================

#include "RxBeamformerMV.h"

#include "USImage.h"
#include "USRawData.h"

#include <utilities/Logging.h>
#include <utilities/cudaUtility.h>
#include <utilities/cublasUtility.h>

using namespace std;

namespace supra
{
	namespace RxBeamformerMV
	{
		template <typename ChannelDataType>
		inline __device__ ChannelDataType 
			readRawData(
				const ChannelDataType* rawData,
				uint32_t sampleIdx, uint32_t channelIdx, uint32_t scanlineIdx,
				uint32_t numSamples, uint32_t numChannels)
		{
			//TEST
		/*	if (sampleIdx >= numSamples || channelIdx >= numChannels)
			{
				printf("readRawData OOB: %d, %d, %d, %d, %d\n", sampleIdx, channelIdx, scanlineIdx,
					numSamples, numChannels);
			}*/
			return rawData[sampleIdx + channelIdx*numSamples + scanlineIdx*numChannels*numSamples];
		}

		template <typename ChannelDataType>
		__device__ uint32_t findFirstValidElement(
			const ChannelDataType* rawData,
			uint32_t sampleIdx, uint32_t scanlineIdx,
			uint32_t numSamples, uint32_t numChannels)
		{
			uint32_t firstValid = 0;
			for (uint32_t channelIdx = 0; channelIdx < numChannels; channelIdx++)
			{
				if (readRawData(rawData, sampleIdx, channelIdx, scanlineIdx, numSamples, numChannels) != 0)
				{
					firstValid = channelIdx;
					break;
				}
			}
			return firstValid;
		}

		template <typename ChannelDataType>
		__device__ uint32_t findLastValidElement(
			const ChannelDataType* rawData,
			uint32_t sampleIdx, uint32_t scanlineIdx,
			uint32_t numSamples, uint32_t numChannels)
		{
			uint32_t lastValid = 0;
			for (int32_t channelIdx = numChannels - 1; channelIdx >= 0; channelIdx--)
			{
				if (readRawData(rawData, sampleIdx, channelIdx, scanlineIdx, numSamples, numChannels) != 0)
				{
					lastValid = channelIdx;
					break;
				}
			}
			return lastValid;
		}

		template <typename ChannelDataType>
		__global__ void computeSubArrayMasks(
			const ChannelDataType* rawData,
			uint32_t numSamples, uint32_t numChannels,
			uint32_t scanlineIdx, uint32_t sampleIdxStart,
			uint32_t subArraySize, uint8_t* subArrayMasks,
			uint32_t* subArraySizes, uint32_t* subArrayOffsets)
		{
			int sampleIdx = blockIdx.x * blockDim.x + threadIdx.x;

			if (sampleIdx < numSamples)
			{
				int numSubArrays = numChannels - subArraySize + 1;

				uint32_t apertureFirst = findFirstValidElement(rawData, sampleIdx, scanlineIdx, numSamples, numChannels);
				uint32_t apertureLast = findLastValidElement(rawData, sampleIdx, scanlineIdx, numSamples, numChannels);

				subArraySizes[sampleIdx] = min(subArraySize, (apertureLast - apertureFirst + 1));

				for (uint32_t subArrayIdx = 0; subArrayIdx < numSubArrays; subArrayIdx++)
				{
					subArrayMasks[subArrayIdx + sampleIdx*numSubArrays] =
						(apertureFirst + subArrayIdx + subArraySizes[sampleIdx] - 1) <= apertureLast;
					subArrayOffsets[subArrayIdx + sampleIdx*numSubArrays] =
						min(max(apertureFirst + subArrayIdx, 0), numChannels - subArraySizes[sampleIdx] + 1);
					if (subArrayIdx > 1)
					{
						subArrayMasks[subArrayIdx + sampleIdx*numSubArrays] =
							subArrayMasks[subArrayIdx + sampleIdx*numSubArrays] &&
							(subArrayOffsets[subArrayIdx + sampleIdx*numSubArrays] !=
								subArrayOffsets[(subArrayIdx - 1) + sampleIdx*numSubArrays]);
					}
				}
			}
		}

		template <typename ChannelDataType>
		__global__ void computeMeansKernel(const ChannelDataType* rawData,
			uint32_t numSamples, uint32_t numChannels, uint32_t scanlineIdx, uint32_t sampleIdxStart,
			uint32_t subArraySize, const uint8_t * subArrayMasks,
			const uint32_t* subArraySizes, const uint32_t* subArrayOffsets, float* means)
		{
			int tIdx = (threadIdx.y * blockDim.x) + threadIdx.x;
			int sampleIdxLocal = (blockIdx.y * gridDim.x) + blockIdx.x;
			int sampleIdx = sampleIdxLocal + sampleIdxStart;

			int numSubArrays = numChannels - subArraySize + 1;

			if (sampleIdx < numSamples)
			{
				int subArraySizeLocal = subArraySizes[sampleIdx];

				for (int vectIdx = tIdx; vectIdx < subArraySizeLocal; vectIdx += blockDim.x*blockDim.y)
				{
					float mean = 0.0f;
					int num = 0;
					for (uint32_t subArrayIdx = 0; subArrayIdx < numSubArrays; subArrayIdx++)
					{
						if (subArrayMasks[subArrayIdx + sampleIdx * numSubArrays])
						{
							auto offset = subArrayOffsets[subArrayIdx + sampleIdx * numSubArrays];
							float x = readRawData(rawData, sampleIdx, offset + vectIdx, scanlineIdx, numSamples, numChannels);
							mean += x;
							num++;
						}
					}

					means[vectIdx + sampleIdxLocal * subArraySize] = mean / static_cast<float>(num);
				}
			}
		}

		template <bool subtractMeans, typename ChannelDataType>
		__global__ void computeRmatrices(const ChannelDataType* rawData,
			uint32_t numSamples, uint32_t numChannels, uint32_t scanlineIdx, uint32_t sampleIdxStart,
			uint32_t subArraySize, const uint8_t * subArrayMasks,
			const uint32_t* subArraySizes, const uint32_t* subArrayOffsets, const float* means, float* Rmatrices)
		{
			int tIdx = (threadIdx.y * blockDim.x) + threadIdx.x;
			int sampleIdxLocal = (blockIdx.y * gridDim.x) + blockIdx.x;
			int sampleIdx = sampleIdxLocal + sampleIdxStart;

			int numSubArrays = numChannels - subArraySize + 1;

			if (sampleIdx < numSamples)
			{
				int subArraySizeLocal = subArraySizes[sampleIdx];
				int numelR = subArraySizeLocal*subArraySizeLocal;
				float* R = &Rmatrices[sampleIdxLocal * subArraySize*subArraySize];

				for (uint32_t subArrayIdx = 0; subArrayIdx < numSubArrays; subArrayIdx++)
				{
					if (subArrayMasks[subArrayIdx + sampleIdx*numSubArrays])
					{
						auto offset = subArrayOffsets[subArrayIdx + sampleIdx*numSubArrays];

						for (int matrixIdx = tIdx; matrixIdx < numelR; matrixIdx += blockDim.x*blockDim.y)
						{
							int colIdx = matrixIdx % subArraySizeLocal;
							int rowIdx = matrixIdx / subArraySizeLocal;

							float xCol = readRawData(rawData, sampleIdx, offset + colIdx, scanlineIdx, numSamples, numChannels);
							float xRow = readRawData(rawData, sampleIdx, offset + rowIdx, scanlineIdx, numSamples, numChannels);
							if (subtractMeans)
							{
								xCol -= means[offset + colIdx + sampleIdx * subArraySize];
								xRow -= means[offset + rowIdx + sampleIdx * subArraySize];
							}

							int matrixStorageIdx = colIdx + rowIdx * subArraySize;

							atomicAdd(&R[matrixStorageIdx], xCol*xRow);
						}
					}
				}
			}
		}

		__global__ void computeTemporalSmoothRmatrices(const float* Rmatrices,
			uint32_t numSamples, uint32_t subArraySize, uint32_t numSubArrays,
			const uint32_t* subArraySizes, uint32_t temporalSmoothing, float* TempRmatrices)
		{
			int tIdx = (threadIdx.y * blockDim.x) + threadIdx.x;
			int sampleIdx = (blockIdx.y * gridDim.x) + blockIdx.x;

			if (sampleIdx < numSamples)
			{
				int subArraySizeLocal = subArraySizes[sampleIdx];
				int numelR = subArraySizeLocal*subArraySizeLocal;
				int numelRfull = subArraySize*subArraySize;

				int firstIdx = max(0, sampleIdx - (int)(temporalSmoothing));
				int lastIdx = min((int)(numSamples)-1, sampleIdx + (int)(temporalSmoothing));

				float scaling = 1.0f;
				for (int matrixIdx = tIdx; matrixIdx < numelR; matrixIdx += blockDim.x*blockDim.y)
				{
					int colIdx = matrixIdx % subArraySizeLocal;
					int rowIdx = matrixIdx / subArraySizeLocal;
					int matrixStorageIdx = colIdx + rowIdx * subArraySize;

					float finalEntry = 0.0f;
					for (int tempIdx = firstIdx; tempIdx <= lastIdx; tempIdx++)
					{
						finalEntry += Rmatrices[matrixStorageIdx + tempIdx*numelRfull];
					}
					TempRmatrices[matrixStorageIdx + sampleIdx*numelRfull] = finalEntry*scaling;
				}
			}
		}

		template <typename T>
		__inline__ __device__ T warpAllReduceSum(T val) {
			for (int mask = warpSize / 2; mask > 0; mask /= 2)
			{
				val += __shfl_xor_sync(0xFFFFFFFF, val, mask);
			}
			return val;
		}

		__global__ void addDiagonalLoading(float* Rmatrices,
			uint32_t numSamples, uint32_t subArraySize, const uint32_t* subArraySizes)
		{
			int tIdx = (threadIdx.y * blockDim.x) + threadIdx.x;
			int sampleIdx = (blockIdx.y * gridDim.x) + blockIdx.x;

			if (sampleIdx < numSamples)
			{
				int subArraySizeLocal = subArraySizes[sampleIdx];
				int numelRfull = subArraySize*subArraySize;

				float* R = &Rmatrices[sampleIdx*numelRfull];

				// compute trace in one block
				float localSum = 0.0f;
				for (int diagIdx = tIdx; diagIdx < subArraySizeLocal; diagIdx += blockDim.x*blockDim.y)
				{
					// subArraySize + 1 (instead of subArraySize) to follow the diagonal
					int matrixIdx = diagIdx * (subArraySize + 1);

					localSum += R[matrixIdx];
				}
				float trace = warpAllReduceSum(localSum);
				float loading = (1.0f / static_cast<float>(subArraySizeLocal)) * trace;

				for (int diagIdx = tIdx; diagIdx < subArraySizeLocal; diagIdx += blockDim.x*blockDim.y)
				{
					// subArraySize + 1 (instead of subArraySize) to follow the diagonal
					int matrixIdx = diagIdx * (subArraySize + 1);

					R[matrixIdx] += loading;
				}

				// Store the loading, s.t. we can use it to fill the diagonal if needed
				if (subArraySize > subArraySizeLocal && tIdx == 0)
				{
					R[subArraySize*subArraySize - 1] = loading;
				}
			}
		}

		__global__ void fillUnusedDiagonal(float* Rmatrices,
			uint32_t numSamples, uint32_t subArraySize, const uint32_t* subArraySizes)
		{
			int tIdx = (threadIdx.y * blockDim.x) + threadIdx.x;
			int sampleIdx = (blockIdx.y * gridDim.x) + blockIdx.x;

			if (sampleIdx < numSamples)
			{
				int subArraySizeLocal = subArraySizes[sampleIdx];
				int numelRfull = subArraySize * subArraySize;

				if (subArraySize > subArraySizeLocal)
				{
					float* R = &Rmatrices[sampleIdx*numelRfull];
					float diagEntry = R[subArraySize*subArraySize - 1];

					for (int diagIdx = subArraySizeLocal + tIdx; diagIdx < subArraySize; diagIdx += blockDim.x*blockDim.y)
					{
						// subArraySize + 1 (instead of subArraySize) to follow the diagonal
						int matrixIdx = diagIdx * (subArraySize + 1);

						R[matrixIdx] = diagEntry;
					}
				}
			}
		}

		template <typename ChannelDataType, typename ImageDataType>
		__global__ void applyWeights(
			const float* RinverseA,
			const float* A,
			const ChannelDataType* rawData,
			uint32_t numSamples,
			uint32_t numChannels,
			uint32_t numScanlines,
			uint32_t scanlineIdx,
			uint32_t sampleIdxStart,
			uint32_t subArraySize,
			float subArrayScalingPower,
			const uint8_t * subArrayMasks,
			const uint32_t * subArraySizes,
			const uint32_t * subArrayOffsets,
			ImageDataType* beamformed)
		{
			int tIdx = (threadIdx.y * blockDim.x) + threadIdx.x;
			int sampleIdxLocal = (blockIdx.y * gridDim.x) + blockIdx.x;
			int sampleIdx = sampleIdxLocal + sampleIdxStart;

			if (sampleIdx < numSamples)
			{
				int numSubArrays = numChannels - subArraySize + 1;
				int subArraySizeLocal = subArraySizes[sampleIdx];

				int numSubArraysActive = 0;
				for (int subArrayIdx = 0; subArrayIdx < numSubArrays; subArrayIdx++)
				{
					if (subArrayMasks[subArrayIdx + sampleIdxLocal*numSubArrays] != 0)
					{
						numSubArraysActive++;
					}
				}

				// compute weight scaling <a, R\a>
				const float* RinvAloc = &RinverseA[sampleIdxLocal * subArraySize];
				const float* Aloc = &A[sampleIdxLocal * subArraySize];
				float weightScaling = 0.0f;
				for (int vectorIdx = tIdx; vectorIdx < subArraySizeLocal; vectorIdx += blockDim.x*blockDim.y)
				{
					weightScaling += RinvAloc[vectorIdx] * Aloc[vectorIdx];
				}
				weightScaling = 1.0f / (warpAllReduceSum(weightScaling) * pow(static_cast<float>(numSubArraysActive), subArrayScalingPower));

				// compute one sample at a time, according to spatial smoothing
				float beamformedSample = 0.0f;
				for (int vectorIdx = tIdx; vectorIdx < subArraySizeLocal; vectorIdx += blockDim.x*blockDim.y)
				{
					float sample = 0.0;
					for (int subArrayIdx = 0; subArrayIdx < numSubArrays; subArrayIdx++)
					{
						if (subArrayMasks[subArrayIdx + sampleIdxLocal*numSubArrays] != 0)
						{
							auto offset = subArrayOffsets[subArrayIdx + sampleIdx*numSubArrays];
							sample += readRawData(rawData, sampleIdx, offset + vectorIdx, scanlineIdx, numSamples, numChannels);
						}
					}
					beamformedSample += sample * RinvAloc[vectorIdx] * weightScaling;
				}
				beamformedSample = warpAllReduceSum(beamformedSample);
				if (tIdx == 0)
				{
					if (abs(beamformedSample) > 1e7 || ::isnan(beamformedSample))
					{
						beamformedSample = 0.0f;
					}
					beamformed[scanlineIdx + sampleIdx * numScanlines] =
						clampCast<ImageDataType>(beamformedSample * numChannels);
				}
			}
		}

		// perform the receive beamforming
		template <typename ChannelDataType, typename ImageDataType>
		shared_ptr<USImage> performRxBeamforming(
			shared_ptr<const USRawData> rawData,
			uint32_t subArraySize,
			uint32_t temporalSmoothing,
			hipblasHandle_t cublasH,
			double subArrayScalingPower,
			bool computeMeans)
		{
			uint32_t sampleBlockSize = 2000;//128;

			//Ensure the raw-data are on the gpu
			auto gRawData = rawData->getData<ChannelDataType>();
			if (!gRawData->isGPU() && !gRawData->isBoth())
			{
				gRawData = std::make_shared<Container<ChannelDataType> >(LocationGpu, *gRawData);
			}
			auto stream = gRawData->getStream();

			uint32_t numScanlines = static_cast<uint32_t>(rawData->getNumScanlines());
			uint32_t numSamples = static_cast<uint32_t>(rawData->getNumSamples());
			uint32_t numChannels = static_cast<uint32_t>(rawData->getNumReceivedChannels());
			if (subArraySize == 0)
			{
				subArraySize = numChannels / 2;
			}

			uint32_t numSubArrays = numChannels - subArraySize + 1;

			size_t numelOut = numScanlines*numSamples;
			shared_ptr<Container<ImageDataType> > pData = std::make_shared<Container<ImageDataType> >(ContainerLocation::LocationGpu, stream, numelOut);

			size_t numelRmatrices = subArraySize*subArraySize* sampleBlockSize;
			shared_ptr<Container<float> > Rmatrices =
				std::make_shared<Container<float> >(ContainerLocation::LocationGpu, stream, numelRmatrices);
			shared_ptr<Container<float> > RmatricesTempSmooth =
				std::make_shared<Container<float> >(ContainerLocation::LocationGpu, stream, numelRmatrices);
			shared_ptr<Container<float> > Avectors =
				std::make_shared<Container<float> >(ContainerLocation::LocationGpu, stream, std::vector<float>(subArraySize*sampleBlockSize, 1.0f));
			shared_ptr<Container<float> > AvectorsOrg =
				std::make_shared<Container<float> >(ContainerLocation::LocationGpu, stream, std::vector<float>(subArraySize*sampleBlockSize, 1.0f));
			shared_ptr<Container<float> > meanVectors =
				std::make_shared<Container<float> >(ContainerLocation::LocationGpu, stream, std::vector<float>(subArraySize*sampleBlockSize, 0.0f));
			shared_ptr<Container<uint8_t> > subArrayMasks =
				std::make_shared<Container<uint8_t> >(ContainerLocation::LocationGpu, stream, numSubArrays*sampleBlockSize);
			shared_ptr<Container<uint32_t> > subArraySizes = 
				std::make_shared<Container<uint32_t> >(ContainerLocation::LocationGpu, stream, sampleBlockSize);
			shared_ptr<Container<uint32_t> > subArrayOffsets =
				std::make_shared<Container<uint32_t> >(ContainerLocation::LocationGpu, stream, numSubArrays*sampleBlockSize);

			shared_ptr<Container<int> > pivotizationArray = 
				std::make_shared<Container<int> >(ContainerLocation::LocationGpu, stream, subArraySize* sampleBlockSize);
			std::vector<int> cublasInfoArrayHost(sampleBlockSize);
			shared_ptr<Container<int> > cublasInfoArrayDevice = std::make_shared<Container<int> >(ContainerLocation::LocationGpu, stream, sampleBlockSize);

			int numelR = subArraySize*subArraySize;
			std::vector<float*> Rpointers(sampleBlockSize);
			std::vector<float*> Apointers(sampleBlockSize);
			for (uint32_t sampleIdx = 0; sampleIdx < sampleBlockSize; sampleIdx++)
			{
				Rpointers[sampleIdx] = RmatricesTempSmooth->get() + sampleIdx* numelR;
				Apointers[sampleIdx] = Avectors->get() + sampleIdx* subArraySize;
			}
			shared_ptr<Container<float*> > RpointersDevice = std::make_shared<Container<float*> >(ContainerLocation::LocationGpu, stream, Rpointers);
			shared_ptr<Container<float*> > ApointersDevice = std::make_shared<Container<float*> >(ContainerLocation::LocationGpu, stream, Apointers);

			for (uint32_t scanlineIdx = 0; scanlineIdx < numScanlines; scanlineIdx++)
			{
				for (uint32_t sampleIdx = 0; sampleIdx < numSamples; sampleIdx += sampleBlockSize)
				{
					uint32_t numSamplesBatch = min(sampleBlockSize, numSamples - sampleIdx);

					cudaSafeCall(hipMemsetAsync(Rmatrices->get(), 0, numelRmatrices * sizeof(float), stream));
					cudaSafeCall(hipMemcpyAsync(Avectors->get(), AvectorsOrg->get(), subArraySize*sampleBlockSize * sizeof(float), hipMemcpyDefault, stream));
					cudaSafeCall(hipMemsetAsync(meanVectors->get(), 0, subArraySize*sampleBlockSize * sizeof(float), stream));
					cudaSafeCall(hipMemsetAsync(subArrayMasks->get(), 0, numSubArrays*sampleBlockSize * sizeof(uint8_t), stream));
					cudaSafeCall(hipMemsetAsync(subArraySizes->get(), 0, sampleBlockSize * sizeof(uint32_t), stream));
					cudaSafeCall(hipMemsetAsync(subArrayOffsets->get(), 0, numSubArrays*sampleBlockSize * sizeof(uint32_t), stream));

					dim3 blockSize(32, 1);
					dim3 gridSize(numSamplesBatch, 1);

					// determine how large the subarrays will be depending on the aperture and which are active
					dim3 gridSizeMasks((numSamplesBatch + blockSize.x - 1) / blockSize.x, 1);
					computeSubArrayMasks << <gridSizeMasks, blockSize, 0, stream >> > (
						gRawData->get(),
						numSamples,
						numChannels,
						scanlineIdx,
						sampleIdx,
						subArraySize,
						subArrayMasks->get(),
						subArraySizes->get(),
						subArrayOffsets->get());
					cudaSafeCall(hipPeekAtLastError());

					if (computeMeans)
					{
						computeMeansKernel << <gridSize, blockSize, 0, stream >> > (
							gRawData->get(),
							numSamples,
							numChannels,
							scanlineIdx,
							sampleIdx,
							subArraySize,
							subArrayMasks->get(),
							subArraySizes->get(),
							subArrayOffsets->get(),
							meanVectors->get());
						cudaSafeCall(hipPeekAtLastError());
					}
				
					// Compute the covariance matrices
					if (computeMeans)
					{
						computeRmatrices<true> << <gridSize, blockSize, 0, stream >> > (
							gRawData->get(),
							numSamples,
							numChannels,
							scanlineIdx,
							sampleIdx,
							subArraySize,
							subArrayMasks->get(),
							subArraySizes->get(),
							subArrayOffsets->get(),
							meanVectors->get(),
							Rmatrices->get());
					}
					else {
						computeRmatrices<false> << <gridSize, blockSize, 0, stream >> > (
							gRawData->get(),
							numSamples,
							numChannels,
							scanlineIdx,
							sampleIdx,
							subArraySize,
							subArrayMasks->get(),
							subArraySizes->get(),
							subArrayOffsets->get(),
							meanVectors->get(),
							Rmatrices->get());
					}
					cudaSafeCall(hipPeekAtLastError());
					
					// Smooth the covariance matrices
					computeTemporalSmoothRmatrices<<<gridSize, blockSize, 0, stream>>> (
						Rmatrices->get(),
						numSamplesBatch,
						subArraySize,
						numSubArrays,
						subArraySizes->get(),
						temporalSmoothing,
						RmatricesTempSmooth->get()
						);
					cudaSafeCall(hipPeekAtLastError());
					
					// Improve condition of matrices
					addDiagonalLoading<<<gridSize, dim3(32, 1), 0, stream>>> (
						RmatricesTempSmooth->get(),
						numSamplesBatch, subArraySize,
						subArraySizes->get());
					cudaSafeCall(hipPeekAtLastError());

					//fill diagonal with ones where necessary
					fillUnusedDiagonal<<<gridSize, dim3(32, 1), 0, stream >>> (
						RmatricesTempSmooth->get(),
						numSamplesBatch, subArraySize,
						subArraySizes->get()
					);
					cudaSafeCall(hipPeekAtLastError());
					//TEST
					cudaSafeCall(hipDeviceSynchronize());

					// solve with cublas
					//TEST
					//auto RmatHost3 = make_shared<Container<float> >(ContainerLocation::LocationHost, *RmatricesTempSmooth);

					cublasSafeCall(hipblasSgetrfBatched(
						cublasH,
						subArraySize,
						(float**)RpointersDevice->get(),
						subArraySize,
						pivotizationArray->get(),
						cublasInfoArrayDevice->get(),
						numSamplesBatch));
					//TEST
					cudaSafeCall(hipDeviceSynchronize());
					auto infoArray = make_shared<Container<int> >(ContainerLocation::LocationHost, *cublasInfoArrayDevice);
					for (size_t k = 0; k < numSamplesBatch; k++)
					{
						assert(infoArray->get()[k] == 0);
					}

					cublasSafeCall(hipblasSgetrsBatched(
						cublasH,
						HIPBLAS_OP_N,
						subArraySize,
						1,
						(const float**)RpointersDevice->get(),
						subArraySize,
						pivotizationArray->get(),
						(float**)ApointersDevice->get(),
						subArraySize,
						cublasInfoArrayHost.data(),
						numSamplesBatch));
					//TEST
					cudaSafeCall(hipDeviceSynchronize());
					for (size_t k = 0; k < numSamplesBatch; k++)
					{
						assert(cublasInfoArrayHost[k] == 0);
					}
				
					// calculate beamforming weights from the solutions and perform beamforming
					applyWeights<<<gridSize, dim3(32, 1), 0, stream>>> (
						Avectors->get(),
						AvectorsOrg->get(),
						gRawData->get(),
						numSamples,
						numChannels,
						numScanlines,
						scanlineIdx,
						sampleIdx,
						subArraySize,
						(float)subArrayScalingPower,
						subArrayMasks->get(),
						subArraySizes->get(),
						subArrayOffsets->get(),
						pData->get()
						);
					cudaSafeCall(hipPeekAtLastError());
				}
			}

			auto retImage = std::make_shared<USImage>(
				vec2s{ numScanlines, numSamples },
				pData,
				rawData->getImageProperties(),
				rawData->getReceiveTimestamp(),
				rawData->getSyncTimestamp());

			return retImage;
		}

		template
			shared_ptr<USImage> performRxBeamforming<int16_t, int16_t>(
				shared_ptr<const USRawData> rawData,
				uint32_t subArraySize,
				uint32_t temporalSmoothing,
				hipblasHandle_t cublasH,
				double subArrayScalingPower,
				bool computeMeans);
		template
			shared_ptr<USImage> performRxBeamforming<int16_t, float>(
				shared_ptr<const USRawData> rawData,
				uint32_t subArraySize,
				uint32_t temporalSmoothing,
				hipblasHandle_t cublasH,
				double subArrayScalingPower,
				bool computeMeans);
		template
			shared_ptr<USImage> performRxBeamforming<float, int16_t>(
				shared_ptr<const USRawData> rawData,
				uint32_t subArraySize,
				uint32_t temporalSmoothing,
				hipblasHandle_t cublasH,
				double subArrayScalingPower,
				bool computeMeans);
		template
			shared_ptr<USImage> performRxBeamforming<float, float>(
				shared_ptr<const USRawData> rawData,
				uint32_t subArraySize,
				uint32_t temporalSmoothing,
				hipblasHandle_t cublasH,
				double subArrayScalingPower,
				bool computeMeans);
	}
}
