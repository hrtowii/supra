#include "hip/hip_runtime.h"
// ================================================================================================
// 
// If not explicitly stated: Copyright (C) 2016, all rights reserved,
//      Rüdiger Göbl 
//		Email r.goebl@tum.de
//      Chair for Computer Aided Medical Procedures
//      Technische Universität München
//      Boltzmannstr. 3, 85748 Garching b. München, Germany
// 
// ================================================================================================

#include "BeamformingMVNode.h"

#include "USImage.h"
#include "USRawData.h"

#include <utilities/Logging.h>
#include <utilities/cudaUtility.h>
//#include <algorithm>
using namespace std;

namespace supra
{
	/// Verifies a cuda call returned "HIPBLAS_STATUS_SUCCESS". Prints error message otherwise.
	/// returns true if no error occured, false otherwise.
	#define cublasSafeCall(_err_) cublasSafeCall2(_err_, __FILE__, __LINE__, FUNCNAME_PORTABLE)

	/// Verifies a cuda call returned "HIPBLAS_STATUS_SUCCESS". Prints error message otherwise.
	/// returns true if no error occured, false otherwise. Calles by cudaSafeCall
	inline bool cublasSafeCall2(hipblasStatus_t err, const char* file, int line, const char* func) {

		//#ifdef CUDA_ERROR_CHECK
		if (HIPBLAS_STATUS_SUCCESS != err) {
			char buf[1024];
			sprintf(buf, "CUBLAS Error (in \"%s\", Line: %d, %s): %d\n", file, line, func, err);
			printf("%s", buf);
			logging::log_error(buf);
			return false;
		}

		//#endif
		return true;
	}

	template <typename ChannelDataType>
	__global__ void computeRmatrices(const ChannelDataType* rawData,
		uint32_t numSamples, uint32_t numChannels, uint32_t scanlineIdx,
		uint32_t subArraySize, float* Rmatrices)
	{
		int tIdx = (threadIdx.y * blockDim.x) + threadIdx.x;
		int sampleIdx = (blockIdx.y * gridDim.x) + blockIdx.x;
		
		if (sampleIdx < numSamples)
		{
			int numSubArrays = numChannels - subArraySize + 1;
			int numelR = subArraySize*subArraySize;
			float* R = &Rmatrices[sampleIdx * numelR];

			for (int subArray = 0; subArray < numSubArrays; subArray++)
			{
				for (int matrixIdx = tIdx; matrixIdx < numelR; matrixIdx += blockDim.x*blockDim.y)
				{
					int colIdx = matrixIdx % subArraySize;
					int rowIdx = matrixIdx / subArraySize;

					float xCol = rawData[sampleIdx + (subArray + colIdx)*numSamples + scanlineIdx*numChannels*numSamples];
					float xRow = rawData[sampleIdx + (subArray + rowIdx)*numSamples + scanlineIdx*numChannels*numSamples];

					atomicAdd(&R[matrixIdx], xCol*xRow);
				}
			}
		}
	}

	__global__ void computeTemporalSmoothRmatrices(const float* Rmatrices,
		uint32_t numSamples, uint32_t subArraySize, uint32_t numSubArrays, 
		uint32_t temporalSmoothing, float* TempRmatrices)
	{
		int tIdx = (threadIdx.y * blockDim.x) + threadIdx.x;
		int sampleIdx = (blockIdx.y * gridDim.x) + blockIdx.x;

		if (sampleIdx < numSamples)
		{
			int numelR = subArraySize*subArraySize;

			int firstIdx = max(0, sampleIdx - temporalSmoothing);
			int lastIdx = min(numSamples - 1, sampleIdx + temporalSmoothing);

			float scaling = 1 / ((lastIdx - firstIdx + 1)*(numSubArrays));
			for (int matrixIdx = tIdx; matrixIdx < numelR; matrixIdx += blockDim.x*blockDim.y)
			{
				float finalEntry = 0.0f;
				for (int tempIdx = firstIdx; tempIdx <= lastIdx; tempIdx++)
				{
					finalEntry += Rmatrices[matrixIdx + tempIdx*numelR];
				}
				//TempRmatrices[matrixIdx + sampleIdx*numelR] = finalEntry*scaling;
				TempRmatrices[0] = finalEntry*scaling;
			}
		}
	}

	__inline__ __device__
	int warpAllReduceSum(int val) {
		for (int mask = warpSize / 2; mask > 0; mask /= 2)
		{
			val += __shfl_xor(val, mask);
		}
		return val;
	}

	__global__ void addDiagonalLoading(float* Rmatrices,
		uint32_t numSamples, uint32_t subArraySize)
	{
		int tIdx = (threadIdx.y * blockDim.x) + threadIdx.x;
		int sampleIdx = (blockIdx.y * gridDim.x) + blockIdx.x;

		if (sampleIdx < numSamples)
		{
			int numelR = subArraySize*subArraySize;

			float* R = &Rmatrices[sampleIdx*numelR];

			// compute trace in one block
			float localSum = 0.0f;
			for (int diagIdx = tIdx; diagIdx < subArraySize; diagIdx += blockDim.x*blockDim.y)
			{
				int matrixIdx = diagIdx * (subArraySize + 1);
				
				localSum += R[matrixIdx];
			}
			float trace = warpAllReduceSum(localSum);
			float loading = (1.0f / static_cast<float>(subArraySize)) * trace;

			for (int diagIdx = tIdx; diagIdx < subArraySize; diagIdx += blockDim.x*blockDim.y)
			{
				int matrixIdx = diagIdx * (subArraySize + 1);

				R[matrixIdx] += loading;
			}
		}
	}

	template <typename ChannelDataType, typename ImageDataType>
	__global__ void applyWeights(
		const float* RinverseA, 
		const float* A, 
		const ChannelDataType* rawData,
		uint32_t numSamples, 
		uint32_t numChannels, 
		uint32_t numScanlines, 
		uint32_t scanlineIdx, 
		uint32_t subArraySize, 
		ImageDataType* beamformed)
	{
		int tIdx = (threadIdx.y * blockDim.x) + threadIdx.x;
		int sampleIdx = (blockIdx.y * gridDim.x) + blockIdx.x;

		if (sampleIdx < numSamples)
		{
			int numSubArrays = numChannels - subArraySize + 1;

			// compute weight scaling <a, R\a>
			const float* RinvAloc = &RinverseA[sampleIdx * subArraySize];
			const float* Aloc = &A[sampleIdx * subArraySize];
			float weightScaling = 0.0f;
			for (int vectorIdx = tIdx; vectorIdx < subArraySize; vectorIdx += blockDim.x*blockDim.y)
			{
				weightScaling += RinvAloc[vectorIdx] * Aloc[vectorIdx];
			}
			weightScaling = 1 / (warpAllReduceSum(weightScaling) * numSubArrays);

			// compute one sample at a time, according to spatial smoothing
			float beamformedSample = 0.0f;
			for (int vectorIdx = tIdx; vectorIdx < subArraySize; vectorIdx += blockDim.x*blockDim.y)
			{
				float sample = 0.0;
				for (int subArray = 0; subArray < numSubArrays; subArray++)
				{
					sample += rawData[sampleIdx + (subArray + vectorIdx)*numSamples + scanlineIdx*numChannels*numSamples];
				}
				beamformedSample += sample*RinvAloc[vectorIdx] * weightScaling;
			}
			beamformedSample = warpAllReduceSum(beamformedSample);
			if (tIdx == 0)
			{
				beamformed[scanlineIdx + sampleIdx * numScanlines] = beamformedSample;
			}
		}
	}

	// perform the receive beamforming
	template <typename ChannelDataType, typename ImageDataType>
	shared_ptr<USImage<ImageDataType> > performRxBeamforming(
		shared_ptr<const USRawData<ChannelDataType> > rawData,
		uint32_t subArraySize,
		uint32_t temporalSmoothing,
		hipblasHandle_t cublasH)
	{
		//Ensure the raw-data are on the gpu
		auto gRawData = rawData->getData();
		if (!rawData->getData()->isGPU() && !rawData->getData()->isBoth())
		{
			gRawData = std::make_shared<Container<ChannelDataType> >(LocationGpu, *gRawData);
		}
		auto stream = gRawData->getStream();

		uint32_t numScanlines = static_cast<uint32_t>(rawData->getNumScanlines());
		uint32_t numSamples   = static_cast<uint32_t>(rawData->getNumSamples());
		uint32_t numChannels  = static_cast<uint32_t>(rawData->getNumReceivedChannels());
		if (subArraySize == 0)
		{
			subArraySize = numChannels;
		}

		uint32_t numSubArrays = numChannels - subArraySize + 1;

		size_t numelOut = numScanlines*numSamples;
		shared_ptr<Container<ImageDataType> > pData = std::make_shared<Container<ImageDataType> >(ContainerLocation::LocationGpu, stream, numelOut);

		size_t numelRmatrices = subArraySize*subArraySize* numSamples;
		shared_ptr<Container<float> > Rmatrices = std::make_shared<Container<float> >(ContainerLocation::LocationGpu, stream, numelRmatrices);
		shared_ptr<Container<float> > RmatricesTempSmooth = std::make_shared<Container<float> >(ContainerLocation::LocationGpu, stream, numelRmatrices);
		shared_ptr<Container<float> > Avectors = std::make_shared<Container<float> >(ContainerLocation::LocationGpu, stream, std::vector<float>(subArraySize*numSamples, 1.0f));
		shared_ptr<Container<float> > AvectorsOrg = std::make_shared<Container<float> >(ContainerLocation::LocationGpu, stream, std::vector<float>(subArraySize*numSamples, 1.0f));

		shared_ptr<Container<int> > pivotizationArray = std::make_shared<Container<int> >(ContainerLocation::LocationGpu, stream, subArraySize* numSamples);
		std::vector<int> cublasInfoArray(numSamples);

		int numelR = subArraySize*subArraySize;
		std::vector<float*> Rpointers(numSamples);
		std::vector<float*> Apointers(numSamples);
		for (uint32_t sampleIdx = 0; sampleIdx < numSamples; sampleIdx++)
		{
			Rpointers[sampleIdx] = RmatricesTempSmooth->get() + sampleIdx* numelR;
			Apointers[sampleIdx] = Avectors->get() + sampleIdx* subArraySize;
		}
		shared_ptr<Container<float*> > RpointersDevice = std::make_shared<Container<float*> >(ContainerLocation::LocationGpu, stream, Rpointers);
		shared_ptr<Container<float*> > ApointersDevice = std::make_shared<Container<float*> >(ContainerLocation::LocationGpu, stream, Apointers);

		for (uint32_t scanlineIdx = 0; scanlineIdx < numScanlines; scanlineIdx++)
		{
			cudaSafeCall(hipMemsetAsync(Rmatrices->get(), 0, numelRmatrices * sizeof(float), stream));
			cudaSafeCall(hipMemcpyAsync(Avectors->get(), AvectorsOrg->get(), subArraySize*numSamples * sizeof(float), hipMemcpyDefault, stream));
			
			//TEST
			cudaSafeCall(hipDeviceSynchronize());

			dim3 blockSize(32, 1);
			dim3 gridSize(numSamples, 1);
			computeRmatrices<<<gridSize, blockSize, 0, stream>>>(
				gRawData->get(),
				numSamples,
				numChannels,
				scanlineIdx,
				subArraySize,
				Rmatrices->get()
			);
			cudaSafeCall(hipPeekAtLastError());
			//TEST
			cudaSafeCall(hipDeviceSynchronize());

			computeTemporalSmoothRmatrices<<<gridSize, blockSize, 0, stream>>>(
				Rmatrices->get(),
				numSamples,
				subArraySize,
				numSubArrays,
				temporalSmoothing,
				Rmatrices->get()//RmatricesTempSmooth->get()
			);
			cudaSafeCall(hipPeekAtLastError());
			//TEST
			cudaSafeCall(hipDeviceSynchronize());

			/*
			addDiagonalLoading<<<gridSize, dim3(32, 1), 0, stream>>>(
				RmatricesTempSmooth->get(),
				numSamples, subArraySize
			);
			cudaSafeCall(hipPeekAtLastError());
			//TEST
			cudaSafeCall(hipDeviceSynchronize());

			cublasSafeCall(hipblasSgetrfBatched(
				cublasH,
				subArraySize,
				(float**)RpointersDevice->get(),
				subArraySize,
				pivotizationArray->get(),
				cublasInfoArray.data(),
				numSamples));
			//TEST
			cudaSafeCall(hipDeviceSynchronize());*/

			/*cublasSafeCall(hipblasSgetrsBatched(
				cublasH,
				HIPBLAS_OP_N,
				subArraySize,
				1,
				(const float**)RpointersDevice->get(),
				subArraySize,
				pivotizationArray->get(),
				(float**)ApointersDevice->get(),
				subArraySize,
				cublasInfoArray.data(),
				numSamples));

			// calculate beamforming weights from that and perform beamforming
			applyWeights<<<gridSize, dim3(32, 1), 0, stream>>>(
				Avectors->get(),
				AvectorsOrg->get(),
				gRawData->get(),
				numSamples,
				numChannels,
				numScanlines,
				scanlineIdx,
				subArraySize,
				pData->get()
			);
			cudaSafeCall(hipPeekAtLastError());*/
		}

		auto retImage = std::make_shared<USImage<ImageDataType> >(
			vec2s{ numScanlines, numSamples },
			pData,
			rawData->getImageProperties(),
			rawData->getReceiveTimestamp(),
			rawData->getSyncTimestamp());

		return retImage;
	}

	BeamformingMVNode::BeamformingMVNode(tbb::flow::graph & graph, const std::string & nodeID)
		: AbstractNode(nodeID)
		, m_node(graph, 1, [this](shared_ptr<RecordObject> inObj) -> shared_ptr<RecordObject> { return checkTypeAndBeamform(inObj); })
		, m_lastSeenImageProperties(nullptr)
	{
		m_callFrequency.setName("BeamformingMV");
		m_valueRangeDictionary.set<uint32_t>("subArraySize", 0, 64, 0, "Sub-array size");
		m_valueRangeDictionary.set<uint32_t>("temporalSmoothing", 0, 10, 3, "temporal smoothing");
		
		configurationChanged();

		cublasSafeCall(hipblasCreate(&m_cublasH));
		cublasSafeCall(hipblasSetAtomicsMode(m_cublasH, HIPBLAS_ATOMICS_ALLOWED));
	}

	BeamformingMVNode::~BeamformingMVNode()
	{
		cublasSafeCall(hipblasDestroy(m_cublasH));
	}

	void BeamformingMVNode::configurationChanged()
	{
		m_subArraySize = m_configurationDictionary.get<uint32_t>("subArraySize");
		m_temporalSmoothing = m_configurationDictionary.get<uint32_t>("temporalSmoothing");
	}

	void BeamformingMVNode::configurationEntryChanged(const std::string& configKey)
	{
		unique_lock<mutex> l(m_mutex);
		if (configKey == "subArraySize")
		{
			m_subArraySize = m_configurationDictionary.get<uint32_t>("subArraySize");
		}
		else if (configKey == "temporalSmoothing")
		{
			m_temporalSmoothing = m_configurationDictionary.get<uint32_t>("temporalSmoothing");
		}
		if (m_lastSeenImageProperties)
		{
			updateImageProperties(m_lastSeenImageProperties);
		}
	}

	shared_ptr<RecordObject> BeamformingMVNode::checkTypeAndBeamform(shared_ptr<RecordObject> inObj)
	{
		unique_lock<mutex> l(m_mutex);

		shared_ptr<USImage<int16_t> > pImageRF = nullptr;
		if (inObj->getType() == TypeUSRawData)
		{
			shared_ptr<const USRawData<int16_t> > pRawData = dynamic_pointer_cast<const USRawData<int16_t>>(inObj);
			if (pRawData)
			{
				if (pRawData->getImageProperties()->getImageState() == USImageProperties::RawDelayed)
				{
					m_callFrequency.measure();

					cublasSafeCall(hipblasSetStream(m_cublasH, pRawData->getData()->getStream()));
					pImageRF = performRxBeamforming<int16_t, int16_t>(
						pRawData, m_subArraySize, m_temporalSmoothing, m_cublasH);
					m_callFrequency.measureEnd();

					if (m_lastSeenImageProperties != pImageRF->getImageProperties())
					{
						updateImageProperties(pImageRF->getImageProperties());
					}
					pImageRF->setImageProperties(m_editedImageProperties);
				}
				else {
					logging::log_error("BeamformingMVNode: Cannot beamform undelayed RawData. Apply RawDelayNode first");
				}
			}
			else {
				logging::log_error("BeamformingMVNode: could not cast object to USRawData type, is it in supported ElementType?");
			}
		}
		return pImageRF;
	}

	void BeamformingMVNode::updateImageProperties(std::shared_ptr<const USImageProperties> imageProperties)
	{
		m_lastSeenImageProperties = imageProperties;
		m_editedImageProperties = make_shared<USImageProperties>(*imageProperties);
		m_editedImageProperties->setImageState(USImageProperties::RF);
		m_editedImageProperties->setSpecificParameter("BeamformingMVNode.subArraySize", m_subArraySize);
		m_editedImageProperties->setSpecificParameter("BeamformingMVNode.temporalSmoothing", m_temporalSmoothing);
	}
}